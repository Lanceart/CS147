
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/
     
    /*************************************************************************/
    // INSERT CODE HERE
	
	int i =  threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n)C[i] = A[i] + B[i];



    /*************************************************************************/
	 
}

void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 256; 

    /*************************************************************************/
    // INSERT CODE HERE
	dim3 dim_grid((n-1)/BLOCK_SIZE + 1,1,1);
	dim3 dim_block(BLOCK_SIZE,1,1);
	VecAdd<<<dim_grid, dim_block>>>(n,A,B,C);
    	
    /*************************************************************************/
}

